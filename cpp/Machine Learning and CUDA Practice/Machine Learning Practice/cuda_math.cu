#include "hip/hip_runtime.h"
#include "cuda_headers.cuh"
#include "cuda_dim.cuh"

__global__
void k_MatAdd(const c_dim3& dim, float *a, float *b) {
	int idx = getGlobalIndex();
	if (idx < dim.prod()) {
		b[idx] = a[idx] + b[idx];
	}
}

__global__
void k_MatDot(const c_dim3& dim, float *a, float *b) {
	int idx = getGlobalIndex();
	if (idx < dim.prod()) {
		b[idx] = a[idx] * b[idx];
	}
}

extern void cuda_MatAdd(const c_dim3& dim, float *a, float *b, float *c) {
	size_t size = dim.prod() * sizeof(float);

	float *d_a; hipMalloc(&d_a, size);
	float *d_b; hipMalloc(&d_b, size);
	c_dim3 *d_dim; hipMalloc(&d_dim, sizeof(c_dim3));

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dim, &dim, sizeof(c_dim3), hipMemcpyHostToDevice);

	c_dim3 threadsPerBlock = dimMin(dim, c_dim3(256,256,256));
	c_dim3 blocksPerGrid = (dim + threadsPerBlock - 1) / threadsPerBlock;
	k_MatAdd <<<blocksPerGrid, threadsPerBlock >>> (*d_dim, d_a, d_b);

	hipMemcpy(c, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}

extern void cuda_MatDot(const c_dim3& dim, float *a, float *b, float *c) {
	size_t size = dim.prod() * sizeof(float);

	float *d_a; hipMalloc(&d_a, size);
	float *d_b; hipMalloc(&d_b, size);
	c_dim3 *d_dim; hipMalloc(&d_dim, sizeof(c_dim3));

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dim, &dim, sizeof(c_dim3), hipMemcpyHostToDevice);

	c_dim3 threadsPerBlock = dimMin(dim, c_dim3(256,256,256));
	c_dim3 blocksPerGrid = (dim + threadsPerBlock - 1) / threadsPerBlock;
	k_MatDot <<<blocksPerGrid, threadsPerBlock >>> (*d_dim, d_a, d_b);

	hipMemcpy(c, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}
