#include <hip/hip_runtime.h>


__global__
void k_VecAdd(int n, float *a, float *b) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < n) {
		b[idx] = a[idx] + b[idx];
	}
}
extern void cuda_VecAdd(int n, float *a, float *b, float *c) {
	size_t size = n * sizeof(float);

	float *d_a;
	hipMalloc(&d_a, size);
	float *d_b;
	hipMalloc(&d_b, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	k_VecAdd <<<blocksPerGrid, threadsPerBlock >>> (n, d_a, d_b);

	hipMemcpy(c, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}